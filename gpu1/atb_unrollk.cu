
#include <hip/hip_runtime.h>
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    // Get the row and column indices of the matrix C element being processed
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Check if the indices are within the bounds of the matrix C
    if (row < Ni && col < Nj)
    {
        double value = 0;
        for (int k = 0; k < Nk; k+=2)
        {
            //value += A[row*Nk + k] * B[k*Nj + col];
            value += A[k*Ni + row] * B[k*Nj + col];
            value += A[(k+1)*Ni + row] * B[(k+1)*Nj + col];
        }
        C[row*Nj + col] = value;
        //C[col*Ni+row] = value;
    }
}
