
#include <hip/hip_runtime.h>
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    // Get the row and column indices of the matrix C element being processed
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Check if the indices are within the bounds of the matrix C
    if (row < Ni && col < Nj)
    {
        double value = 0;
        for (int k = 0; k < Nk; k++)
        {
            //value += A[row*Nk + k] * B[k*Nj + col];
            value += A[k*Ni + row] * B[k*Nj + col];
        }
        C[row*Nj + col] = value;
        //C[col*Ni+row] = value;
    }
}