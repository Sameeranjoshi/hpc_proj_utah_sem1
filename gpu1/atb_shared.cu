#include "hip/hip_runtime.h"
_global_ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk)
{   int BLOCK_SIZE = 8;
    _shared_ double mat_1_tile[8][8];
    _shared_ double mat_2_tile[8][8];

    double acc_sum{0};
    int temp = ceil((double)Nk / BLOCK_SIZE);
    for (int tile_idx = 0; tile_idx < temp; ++tile_idx)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = tile_idx * blockDim.x + threadIdx.x;
        if ((i < Ni) && (j < Nk))
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = A[j * Ni + i];
            //mat_1_tile[threadIdx.x][threadIdx.y] = A[i * Nk + j];
        }
        else
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = 0;
        }
        i = tile_idx * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if ((i < Nk) && (j < Nj))
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = B[i * Nj + j];
        }
        else
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            acc_sum += mat_1_tile[threadIdx.y][k] * mat_2_tile[k][threadIdx.x];
        }
        __syncthreads();
    }

    // 2D block and 2D thread
    // Each thread computes one cell in C.
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i < Ni) && (j < Nj))
    {
        C[i * Nj + j] = acc_sum;
    }
}
