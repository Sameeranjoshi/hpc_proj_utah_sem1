
#include <hip/hip_runtime.h>
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    // Get the row and column indices of the matrix C element being processed
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Check if the indices are within the bounds of the matrix C
    if (row < Ni && col < Nj)
    {
        double value = 0;
        double value1 = 0;
        for (int k = 0; k < Nk; k++)
        {
            //value += A[row*Nk + k] * B[k*Nj + col];
            value += A[k*Ni + row] * B[k*Nj + col];
            value1 += A[k*Ni + row] * B[k*(Nj+1) + col];
    
        }
        C[row*Nj + col] = value;
        C[row*(Nj+1) + col] = value1;
        //C[col*Ni+row] = value;
    }
}
