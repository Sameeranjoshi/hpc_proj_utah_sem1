
#include <hip/hip_runtime.h>
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    // Get the row and column indices of the matrix C element being processed
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Check if the indices are within the bounds of the matrix C
    if (row < Ni && col < Nj)
    {
        double value = 0;
        for (int k = 0; k < Nk; k+=2)
        {
            //value += A[row*Nk + k] * B[k*Nj + col];
            value += A[k*Ni + row] * B[col*Nk + k];
            value += A[(k+1)*Ni + row] * B[col*Nk + (k+1)];
        }
        C[row*Nj + col] = value;
        //C[col*Ni+row] = value;
    }
}
