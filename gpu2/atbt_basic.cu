
#include <hip/hip_runtime.h>
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    // Get the row and column indices of the matrix C element being processed
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Check if the indices are within the bounds of the matrix C
    if (row < Ni && col < Nj)
    {
        double value = 0;
        for (int k = 0; k < Nk; k++)
        {
            //value += A[row*Nk + k] * B[k*Nj + col];
            value += A[k*Ni + row] * B[col*Nk + k];
        }
        C[row*Nj + col] = value;
        //C[col*Ni+row] = value;
    }
}