
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define threshold 0.0000001

#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4

void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);

int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k,Ni,Nj,Nk;

  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);
  h_A = (double *) malloc(sizeof(double)*Ni*Nk);
  h_B = (double *) malloc(sizeof(double)*Nk*Nj);
  h_C = (double *) malloc(sizeof(double)*Ni*Nj);
  h_Cref = (double *) malloc(sizeof(double)*Ni*Nj);
  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = k*Ni+i-1;
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[j*Nk+k] = j*Nk+k+1;
  for (i=0; i<Ni; i++)
   for (j=0; j<Nj; j++) {
    h_C[i*Nj+j] = 0;
    h_Cref[i*Nj+j] = 0;}

  for (i=0;i<Ni;i++)
   for (k=0;k<Nk;k++)
    for (j=0;j<Nj;j++)
      //h_Cref[i][j] += h_A[k][i]*h_B[j][k];
     h_Cref[i*Nj+j] += h_A[i+Ni*k]*h_B[k+Nk*j];
     //h_Cref[i*Nj+j] += h_A[i*Nk+k]*h_B[k*Nj+j];
  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Nk*Ni*sizeof(double));
  hipMalloc(&d_B, Nj*Nk*sizeof(double));
  hipMalloc(&d_C, Ni*Nj*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Nk*Ni*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nj*Nk*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  int blocksize = 8;
  dim3 block(blocksize,blocksize);  
  dim3 grid((Nj+blocksize-1)/blocksize,(Ni+blocksize-1)/blocksize);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  for(int trial=0;trial<5;trial++)
  {
   hipEventRecord(start);
   // Launch kernel
   atbt<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);
   hipEventRecord(stop);
   checkCUDAError("kernel launch");
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
//   cudaDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, Ni*Nj*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int l = 0; l < Ni*Nj; l++) if (fabs((h_C[l] - h_Cref[l])/h_Cref[l])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", l, h_C[l], h_Cref[l]); return -1;}
   printf("<Ni=%d,Nj=%d,Nk=%d>: Trial %d: GFLOPS: %.2f\n",Ni,Nj,Nk,trial,2.0e-6*Ni*Nj*Nk/elapsedTime);
  }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


